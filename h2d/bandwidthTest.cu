


#include <hip/hip_runtime.h>
#include <thread>
#include <cstddef>
#include <stdlib.h>


__global__ void emptyKernel(float* d_data)
{
  // 不执行任何操作
}


void bandwidthTest(float* h_data, float* d_data, int size, hipStream_t stream)
{
  // 将数据从主机复制到设备
  hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream);

  // 运行空的核函数
  emptyKernel<<<1, 1, 0, stream>>>(d_data);

  // 确保所有操作都完成
  hipStreamSynchronize(stream);
}

int main(int argc, char* argv[])
{
  size_t size = static_cast<size_t>(1024) * 1024 * 1024; // 1GB

  // 分配主机内存
  float* h_data;
  hipHostMalloc((void**)&h_data, size, hipHostMallocDefault);

  // 初始化数据
  for (size_t i = 0; i < size / sizeof(float); ++i)
  {
    h_data[i] = static_cast<float>(i);
  }

  //测试GPU卡的数量
  long device_num= strtol(argv[1], NULL, 10);

  // 创建 CUDA 流
  hipStream_t streams[device_num];
  for(int i = 0; i < device_num; i++) {
    hipStreamCreate(&streams[i]);
  }

  // 分配设备内存
  float *d_data[device_num];
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipMalloc((void**)&d_data[i], size);
  }

  // 创建 CUDA 事件来记录开始和结束时间
  hipEvent_t start[device_num], stop[device_num];
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipEventCreate(&start[i]);
    hipEventCreate(&stop[i]);
  }

  // 预热 GPU
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    bandwidthTest(h_data, d_data[i], size, streams[i]);
  }

  // 等待所有预热操作完成
  for(int i = 0; i < device_num; i++) {
    hipStreamSynchronize(streams[i]);
  }

  // 清空 GPU 的缓存
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipMemset(d_data[i], 0, size);
  }

  // 记录开始时间
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipEventRecord(start[i], streams[i]);
  }

  // 运行带宽测试
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    bandwidthTest(h_data, d_data[i], size, streams[i]);
  }

  // 记录结束时间
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipEventRecord(stop[i], streams[i]);
  }

  // 等待所有操作完成
  for(int i = 0; i < device_num; i++) {
    hipStreamSynchronize(streams[i]);
  }

  // 计算所用时间
  float milliseconds = 0;
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    float temp = 0;
    hipEventElapsedTime(&temp, start[i], stop[i]);
    milliseconds = std::max(milliseconds, temp);
  }

  // 计算带宽（GB/s）
  float bandwidth = device_num * size / milliseconds / 1e6; // size * device_num (for eight GPUs)

  // 打印带宽
  printf("Total bandwidth: %.2f GB/s\n", bandwidth);

  // 销毁 CUDA 事件
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipEventDestroy(start[i]);
    hipEventDestroy(stop[i]);
  }

  // 释放内存
  hipHostFree(h_data);
  for(int i = 0; i < device_num; i++) {
    hipSetDevice(i);
    hipFree(d_data[i]);
  }

  // 销毁 CUDA 流
  for(int i = 0; i < device_num; i++) {
    hipStreamDestroy(streams[i]);
  }

  return 0;
}

