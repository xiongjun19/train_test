
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <vector>
#include <omp.h>
#include <chrono>


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

void profileCopies(float        *h_a, 
                   float        *h_b, 
                   float        *d, 
                   unsigned int  n,
                   char         *desc)
{
  printf("\n%s transfers\n", desc);

  unsigned int bytes = n * sizeof(float);

  // events for timing
  hipEvent_t startEvent, stopEvent; 

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  float time;
  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  for (int i = 0; i < n; ++i) {
    if (h_a[i] != h_b[i]) {
      printf("*** %s transfers failed ***\n", desc);
      break;
    }
  }

  // clean up events
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
}


int main(int argc, char* argv[])
{
  long nElements = strtol(argv[1], NULL, 10);
  const unsigned int bytes = nElements * sizeof(float);
  long deviceNum= strtol(argv[2], NULL, 10);
  std::cout << " input device Num is: " << deviceNum << std::endl;
  std::cout << " input bytes  is: " << bytes << std::endl;
  std::vector<float * > memVec;
  for(int i=0; i < deviceNum; ++i){
      float * hPinned;  
      hipHostMalloc((void**)&hPinned, bytes, hipHostMallocDefault);
      memVec.push_back(hPinned);
      for(int j=0; j<nElements; ++j){
	      memVec[i][j] = i+j;
      }
  }
  // then init device Mem
  std::vector<float * > devVec;
  for(int i=0; i < deviceNum; ++i){
      hipSetDevice(i);
      float * dMem;
      hipMalloc((void**)&dMem, bytes);
      devVec.push_back(dMem);
  }
  


  std::chrono::steady_clock::time_point beg = std::chrono::steady_clock::now(); 
  #pragma omp parallel for num_threads(deviceNum)
  for(int i=0; i<deviceNum; ++i){
      hipSetDevice(i);
      hipMemcpy(devVec[i], memVec[i], bytes, hipMemcpyHostToDevice);
      hipDeviceSynchronize();
  }
  std::chrono::steady_clock::time_point end  = std::chrono::steady_clock::now(); 
  auto timeDiff = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();
  auto bandWidth =  deviceNum * bytes * 1e-3 / timeDiff;
  std::cout << "time consumed: " << timeDiff << "\t Host to Device bandwidth (GB/s): " << bandWidth << std::endl;
  for(int i=0; i < deviceNum; ++i){
      hipFree(devVec[i]);
      hipHostFree(memVec[i]);
  }

  return 0;
}
